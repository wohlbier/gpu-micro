#include "hip/hip_runtime.h"
#include <getopt.h>
#include <stdio.h>

#include "common.h"

thread_local int is_main_thread = 0;

static int nThreads = 1;
static int nGpus = 1;
static size_t minBytes = 32*1024*1024;
static size_t maxBytes = 32*1024*1024;
static size_t stepBytes = 1*1024*1024;
static size_t stepFactor = 1;
static int datacheck = 1;
static int warmup_iters = 5;
static int iters = 20;

double parsesize(char *value) {
    long long int units;
    double size;

    if (strchr(value, 'G') != NULL) {
        units=1024*1024*1024;
    } else if (strchr(value, 'M') != NULL) {
	units=1024*1024;
    } else if (strchr(value, 'K') != NULL) {
        units=1024;
    } else {
        units=1;
    }

    size = atof(value)*units;
    return size;
}

testResult_t AllocateBuffs(void **sendbuff, size_t sendBytes, void **recvbuff,
                           size_t recvBytes, void **expected, size_t nbytes,
                           int nranks) {
    printf("%d %d\n", nbytes, recvBytes);

    CUDACHECK(hipMalloc(sendbuff, nbytes));
    CUDACHECK(hipMalloc(recvbuff, nbytes));
    CUDACHECK(hipMalloc(expected, recvBytes));
    return testSuccess;
}

testResult_t run();

int main(int argc, char* argv[]) {
  // Make sure everyline is flushed so that we see the progress of the test
  setlinebuf(stdout);

  // Parse args
  int longindex;
  static struct option longopts[] = {
      {"nthreads", required_argument, 0, 't'},
      {"ngpus", required_argument, 0, 'g'},
      {"minbytes", required_argument, 0, 'b'},
      {"maxbytes", required_argument, 0, 'e'},
      {"stepbytes", required_argument, 0, 'i'},
      {"stepfactor", required_argument, 0, 'f'},
      {"iters", required_argument, 0, 'n'},
      {"agg_iters", required_argument, 0, 'm'},
      {"warmup_iters", required_argument, 0, 'w'},
      {"check", required_argument, 0, 'c'},
      {"help", no_argument, 0, 'h'}
  };

  while(1) {
      int c;
      c = getopt_long(argc, argv, "t:g:b:e:i:f:n:w:c:h", longopts,
                      &longindex);
      if (c == -1)
          break;
      switch(c) {
      case 't':
          nThreads = strtol(optarg, NULL, 0);
          break;
      case 'g':
          nGpus = strtol(optarg, NULL, 0);
          break;
      case 'b':
          minBytes = (size_t)parsesize(optarg);
          break;
      case 'e':
          maxBytes = (size_t)parsesize(optarg);
          break;
      case 'i':
          stepBytes = strtol(optarg, NULL, 0);
          break;
      case 'f':
          stepFactor = strtol(optarg, NULL, 0);
          break;
      case 'n':
          iters = (int)strtol(optarg, NULL, 0);
          break;
      case 'w':
          warmup_iters = (int)strtol(optarg, NULL, 0);
          break;
      case 'c':
          datacheck = (int)strtol(optarg, NULL, 0);
          break;
      case 'h':
          printf("USAGE: %s \n\t"
                 "[-t,--nthreads <num threads>] \n\t"
                 "[-g,--ngpus <gpus per thread>] \n\t"
                 "[-b,--minbytes <min size in bytes>] \n\t"
                 "[-e,--maxbytes <max size in bytes>] \n\t"
                 "[-i,--stepbytes <increment size>] \n\t"
                 "[-f,--stepfactor <increment factor>] \n\t"
                 "[-n,--iters <iteration count>] \n\t"
                 "[-m,--agg_iters <aggregated iteration count>] \n\t"
                 "[-w,--warmup_iters <warmup iteration count>] \n\t"
                 "[-c,--check <0/1>] \n\t"
                 "[-h,--help]\n",
                 basename(argv[0]));
        return 0;
      default:
          printf("invalid option \n");
          printf("USAGE: %s \n\t"
                 "[-t,--nthreads <num threads>] \n\t"
                 "[-g,--ngpus <gpus per thread>] \n\t"
                 "[-b,--minbytes <min size in bytes>] \n\t"
                 "[-e,--maxbytes <max size in bytes>] \n\t"
                 "[-i,--stepbytes <increment size>] \n\t"
                 "[-f,--stepfactor <increment factor>] \n\t"
                 "[-n,--iters <iteration count>] \n\t"
                 "[-m,--agg_iters <aggregated iteration count>] \n\t"
                 "[-w,--warmup_iters <warmup iteration count>] \n\t"
                 "[-c,--check <0/1>] \n\t"
                 "[-h,--help]\n",
                 basename(argv[0]));
          return 0;
    }
  }
  return run();
}

testResult_t run() {
    int nProcs = 1, proc = 0;
    int localRank = 0;
    char hostname[1024];
    getHostName(hostname, 1024);

    is_main_thread = (proc == 0) ? 1 : 0;

    PRINT("# nThread %d nGpus %d minBytes %ld maxBytes %ld step: %ld(%s) warmup iters: %d iters: %d validation: %d \n",
          nThreads, nGpus, minBytes, maxBytes,
          (stepFactor > 1)?stepFactor:stepBytes,
          (stepFactor > 1)?"factor":"bytes",
          warmup_iters, iters, datacheck);

    PRINT("# Using devices\n");
#define MAX_LINE 2048
    char line[MAX_LINE];
    int len = 0;
    for (int i=0; i<nThreads*nGpus; i++) {
        int cudaDev = localRank*nThreads*nGpus+i;
        int rank = proc*nThreads*nGpus+i;
        hipDeviceProp_t prop;
        CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
        len += snprintf(line+len, MAX_LINE-len,
                        "#Rank %2d Pid %6d on %10s device %2d [0x%02x] %s\n",
                        rank, getpid(), hostname, cudaDev, prop.pciBusID,
                        prop.name);
    }
    PRINT("%s", line);

    hipStream_t streams[nGpus*nThreads];
    void* sendbuffs[nGpus*nThreads];
    void* recvbuffs[nGpus*nThreads];
    void* expected[nGpus*nThreads];
    size_t sendBytes, recvBytes;

    microTestEngine.getBuffSize(&sendBytes, &recvBytes, (size_t)maxBytes,
                                (size_t)nProcs*nGpus*nThreads);

  for (int i=0; i<nGpus*nThreads; i++) {
      CUDACHECK(hipSetDevice(localRank*nThreads*nGpus+i));
      AllocateBuffs(sendbuffs+i, sendBytes, recvbuffs+i, recvBytes, expected+i,
                    (size_t)maxBytes, nProcs*nThreads*nGpus);
      CUDACHECK(hipStreamCreateWithFlags(streams+i, hipStreamNonBlocking));
  }

//  //if parallel init is not selected, use main thread to initialize NCCL
//  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nThreads*nGpus);
//  if (!parallel_init) {
//     if (nProcs == 1) {
//       int gpuArray[nGpus*nThreads];
//       for (int i=0; i<nGpus*nThreads; i++) gpuArray[i] = i;
//       NCCLCHECK(ncclCommInitAll(comms, nGpus*nThreads, gpuArray));
//     } else {
//       NCCLCHECK(ncclGroupStart());
//       for (int i=0; i<nGpus*nThreads; i++) {
//         CUDACHECK(hipSetDevice(localRank*nThreads*nGpus+i));
//         NCCLCHECK(ncclCommInitRank(comms+i, nProcs*nThreads*nGpus, ncclId, pro\
//c*nThreads*nGpus+i));
//       }
//       NCCLCHECK(ncclGroupEnd());
//     }
//  }

//  int errors[nThreads];
//  double bw[nThreads];
//  double* delta;
//  CUDACHECK(hipHostAlloc(&delta, sizeof(double)*nThreads, cudaHostAllocPortabl\
//e | hipHostMallocMapped));
//  int bw_count[nThreads];
//  for (int t=0; t<nThreads; t++) {
//    bw[t] = 0.0;
//    errors[t] = bw_count[t] = 0;
//  }
//  PRINT("#\n");
//  print_header();

//  int* sync = (int*)calloc(2, sizeof(int));
//  int* barrier = (int*)calloc(2, sizeof(int));

//  struct testThread threads[nThreads];
//  memset(threads, 0, sizeof(struct testThread)*nThreads);

//  for (int t=nThreads-1; t>=0; t--) {
//    threads[t].args.minbytes=minBytes;
//    threads[t].args.maxbytes=maxBytes;
//    threads[t].args.stepbytes=stepBytes;
//    threads[t].args.stepfactor=stepFactor;
//    threads[t].args.localRank = localRank;

//    threads[t].args.nProcs=nProcs;
//    threads[t].args.proc=proc;
//    threads[t].args.nThreads=nThreads;
//    threads[t].args.thread=t;
//    threads[t].args.nGpus=nGpus;
//    threads[t].args.sendbuffs = sendbuffs+t*nGpus;
//    threads[t].args.recvbuffs = recvbuffs+t*nGpus;
//    threads[t].args.expected = expected+t*nGpus;
//    threads[t].args.ncclId = ncclId;
//    threads[t].args.comms=comms+t*nGpus;
//    threads[t].args.streams=streams+t*nGpus;

//    threads[t].args.barrier = (volatile int*)barrier;
//    threads[t].args.barrier_idx = 0;
//    threads[t].args.sync = (volatile int*)sync;
//    threads[t].args.sync_idx = 0;
//    threads[t].args.deltaThreads = delta;
//    threads[t].args.deltaHost = (delta + t);
//    threads[t].args.delta = delta;
//    threads[t].args.errors=errors+t;
//    threads[t].args.bw=bw+t;
//    threads[t].args.bw_count=bw_count+t;

//    threads[t].func = parallel_init ? threadInit : threadRunTests;
//    if (t)
//      TESTCHECK(threadLaunch(threads+t));
//    else
//      TESTCHECK(threads[t].func(&threads[t].args));
//  }

//  // Wait for other threads and accumulate stats and errors
//  for (int t=nThreads-1; t>=0; t--) {
//    if (t) pthread_join(threads[t].thread, NULL);
//    TESTCHECK(threads[t].ret);
//    if (t) {
//      errors[0] += errors[t];
//      bw[0] += bw[t];
//      bw_count[0] += bw_count[t];
//    }
//  }

//#ifdef MPI_SUPPORT
//  MPI_Allreduce(MPI_IN_PLACE, &errors[0], 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
//#endif

// if (!parallel_init) {
//    for(int i=0; i<nGpus*nThreads; ++i)
//      NCCLCHECK(ncclCommDestroy(comms[i]));
//    free(comms);
//  }

//  // Free off CUDA allocated memory
//  for (int i=0; i<nGpus*nThreads; i++) {
//    CUDACHECK(hipFree(sendbuffs[i]));
//    CUDACHECK(hipFree(recvbuffs[i]));
//    CUDACHECK(hipFree(expected[i]));
//  }
//  CUDACHECK(hipHostFree(delta));

//  char* str = getenv("NCCL_TESTS_MIN_BW");
//  double check_avg_bw = str ? atof(str) : -1;
//  bw[0] /= bw_count[0];

//  PRINT("# Out of bounds values : %d %s\n", errors[0], errors[0] ? "FAILED" : "\
//OK");
//  PRINT("# Avg bus bandwidth    : %g %s\n", bw[0], check_avg_bw == -1 ? "" : (b\
//w[0] < check_avg_bw*(0.9) ? "FAILED" : "OK"));
//  PRINT("#\n");
//#ifdef MPI_SUPPORT
//  MPI_Finalize();
//#endif

//  // 'cuda-memcheck --leak-check full' requires this
//  hipDeviceReset();

//  if (errors[0] || bw[0] < check_avg_bw*(0.9))
//    exit(EXIT_FAILURE);
//  else
//    exit(EXIT_SUCCESS);
    testResult_t tr;
    return tr;
}
